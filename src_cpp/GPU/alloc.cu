#include "cuda_helpers.h"

void host_mem_alloc(char *&host_buf, size_t size_to_alloc)
{
    CUDA_CALL(hipHostAlloc(&host_buf, size_to_alloc, hipHostMallocPortable))
    return;
}

void host_mem_free(char *&host_buf)
{
    CUDA_CALL(hipHostFree(host_buf));
    return;
}

void device_mem_alloc(char *&device_buf, size_t size_to_alloc)
{
    CUDA_CALL(hipMalloc(&device_buf, size_to_alloc));
    CUDA_CALL(hipMemset(device_buf, 0, size_to_alloc));
    return;
}

void device_mem_free(char *&device_buf)
{
    if (device_buf) {
        CUDA_CALL(hipFree(device_buf));
    }
    return;
}

