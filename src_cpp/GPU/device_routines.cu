#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda_helpers.h"
#include "device_routines.h"
#include <hip/hip_runtime.h>
#include <assert.h>

static hipStream_t stream_main = 0, stream_workload = 0;
static hipEvent_t event = 0;
static bool initialized = false;

void device_init_contexts()
{
    CUDA_CALL(hipStreamCreateWithFlags(&stream_main, hipStreamNonBlocking))
    CUDA_CALL(hipStreamCreateWithFlags(&stream_workload, hipStreamNonBlocking))
    CUDA_CALL(hipEventCreateWithFlags(&event, hipEventDisableTiming));
}

void device_sync_context()
{
    CUDA_CALL(hipStreamSynchronize(stream_main));
    CUDA_CALL(hipStreamSynchronize(stream_workload));
}

size_t device_get_num_of_dev()
{
    int n;
    CUDA_CALL(hipGetDeviceCount(&n));
    return n;
}

void device_set_current(size_t n)
{
    assert(!initialized);
    std::cout << "GPU device set: cuda_id=" << n << std::endl;
    CUDA_CALL(hipSetDevice(n));
    device_init_contexts();
    initialized = true;
}

void device_set_current(const std::string &pci_id)
{
    assert(!initialized);
    hipDevice_t dev;
    char devname[256];
    CUDADRIVER_CALL(hipInit(0));
    CUDADRIVER_CALL(hipDeviceGetByPCIBusId(&dev, pci_id.c_str()));
    CUDADRIVER_CALL(hipDeviceGetName(devname, 256, dev));
    std::cout << "GPU device set: pci_id=" << pci_id << ", name=" << devname << " (with hwloc)" << std::endl;
    initialized = true;
}

bool device_is_idle()
{
    if (event) {
        CUDA_CALL(hipEventRecord(event, stream_workload));
        hipError_t ret = hipEventQuery(event);
        if (ret != hipErrorNotReady && ret != hipSuccess) {
            // error case: throw exception
            CUDA_CALL(ret);
        }
        if (ret == hipErrorNotReady) {
            // stream has some load currently, not idle
            return false;
        }
    }
    return true;
}

template <int SIZE>
__global__ void workload(int ncycles, int CALIBRATION_CONST) {
    __shared__ double a[SIZE][SIZE], b[SIZE][SIZE], c[SIZE][SIZE];
    while (ncycles--) {
        for (int N = 0; N < CALIBRATION_CONST; N++) {
            for (int i = 0; i < SIZE; i++) {
                for (int j = 0; j < SIZE; j++) {
                    for (int k = 0; k < SIZE; k++) {
                        c[i][j] += a[i][k] * b[k][j] + N * N;
                    }
                }
            }
        }
    }
}

void device_submit_workload(int ncycles, int calibration_const)
{
    constexpr int array_dim = 10;
    workload<array_dim><<<1, 1, 0, stream_workload>>>(ncycles, calibration_const);
}

void d2h_transfer(char *to, char *from, size_t size, transfer_t type)
{
    CUDA_CALL(hipMemcpyAsync(to, from, size, hipMemcpyDeviceToHost, 
                              type == transfer_t::MAIN ? stream_main : stream_workload));
    if (type == transfer_t::MAIN) {
        CUDA_CALL(hipStreamSynchronize(stream_main))
    }
}

void h2d_transfer(char *to, char *from, size_t size, transfer_t type)
{
    CUDA_CALL(hipMemcpyAsync(to, from, size, hipMemcpyHostToDevice, 
                              type == transfer_t::MAIN ? stream_main : stream_workload));
    if (type == transfer_t::MAIN) {
        CUDA_CALL(hipStreamSynchronize(stream_main))
    }
}
